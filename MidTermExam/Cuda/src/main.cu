#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#include "desKeyGenerator.h"
#include "desTextEncryptor.h"
#include "utility.h"

using namespace std;

__device__ const char d_allowed_char [] = {'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h',
'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p',
'q', 'r', 's', 't', 'u', 'v', 'w', 'x',
'y', 'z', 'A', 'B', 'C', 'D', 'E', 'F',
'G', 'H', 'I', 'J', 'K', 'L', 'M', 'N',
'O', 'P', 'Q', 'R', 'S', 'T', 'U', 'V',
'W', 'X', 'Y', 'Z', '1', '2', '3', '4',
'5', '6', '7', '8', '9', '0', '.', '/'};
__device__ long d_allowed_char_size = sizeof(d_allowed_char) / sizeof(d_allowed_char[0]);

__device__ int d_initial_permutation_table[64] = {
        58, 50, 42, 34, 26, 18, 10, 2,
        60, 52, 44, 36, 28, 20, 12, 4,
        62, 54, 46, 38, 30, 22, 14, 6,
        64, 56, 48, 40, 32, 24, 16, 8,
        57, 49, 41, 33, 25, 17,  9, 1,
        59, 51, 43, 35, 27, 19, 11, 3,
        61, 53, 45, 37, 29, 21, 13, 5,
        63, 55, 47, 39, 31, 23, 15, 7
};

__device__ int d_expansion_permutation_table[48] = {
        32, 1, 2, 3, 4, 5, 4, 5,
        6, 7, 8, 9, 8, 9, 10, 11,
        12, 13, 12, 13, 14, 15, 16, 17,
        16, 17, 18, 19, 20, 21, 20, 21,
        22, 23, 24, 25, 24, 25, 26, 27,
        28, 29, 28, 29, 30, 31, 32, 1
};

__device__ int d_s_box_table[8][4][16] = {
        {
                14, 4, 13, 1, 2, 15, 11, 8, 3, 10, 6, 12, 5, 9, 0, 7,
                0, 15, 7, 4, 14, 2, 13, 1, 10, 6, 12, 11, 9, 5, 3, 8,
                4, 1, 14, 8, 13, 6, 2, 11, 15, 12, 9, 7, 3, 10, 5, 0,
                15, 12, 8, 2, 4, 9, 1, 7, 5, 11, 3, 14, 10, 0, 6, 13
        },
        {
                15, 1, 8, 14, 6, 11, 3, 4, 9, 7, 2, 13, 12, 0, 5, 10,
                3, 13, 4, 7, 15, 2, 8, 14, 12, 0, 1, 10, 6, 9, 11, 5,
                0, 14, 7, 11, 10, 4, 13, 1, 5, 8, 12, 6, 9, 3, 2, 15,
                13, 8, 10, 1, 3, 15, 4, 2, 11, 6, 7, 12, 0, 5, 14, 9
        },
        {
                10, 0, 9, 14, 6, 3, 15, 5, 1, 13, 12, 7, 11, 4, 2, 8,
                13, 7, 0, 9, 3, 4, 6, 10, 2, 8, 5, 14, 12, 11, 15, 1,
                13, 6, 4, 9, 8, 15, 3, 0, 11, 1, 2, 12, 5, 10, 14, 7,
                1, 10, 13, 0, 6, 9, 8, 7, 4, 15, 14, 3, 11, 5, 2, 12
        },
        {
                7, 13, 14, 3, 0, 6, 9, 10, 1, 2, 8, 5, 11, 12, 4, 15,
                13, 8, 11, 5, 6, 15, 0, 3, 4, 7, 2, 12, 1, 10, 14, 9,
                10, 6, 9, 0, 12, 11, 7, 13, 15, 1, 3, 14, 5, 2, 8, 4,
                3, 15, 0, 6, 10, 1, 13, 8, 9, 4, 5, 11, 12, 7, 2, 14
        },
        {
                2, 12, 4, 1, 7, 10, 11, 6, 8, 5, 3, 15, 13, 0, 14, 9,
                14, 11, 2, 12, 4, 7, 13, 1, 5, 0, 15, 10, 3, 9, 8, 6,
                4, 2, 1, 11, 10, 13, 7, 8, 15, 9, 12, 5, 6, 3, 0, 14,
                11, 8, 12, 7, 1, 14, 2, 13, 6, 15, 0, 9, 10, 4, 5, 3
        },
        {
                12, 1, 10, 15, 9, 2, 6, 8, 0, 13, 3, 4, 14, 7, 5, 11,
                10, 15, 4, 2, 7, 12, 9, 5, 6, 1, 13, 14, 0, 11, 3, 8,
                9, 14, 15, 5, 2, 8, 12, 3, 7, 0, 4, 10, 1, 13, 11, 6,
                4, 3, 2, 12, 9, 5, 15, 10, 11, 14, 1, 7, 6, 0, 8, 13
        },
        {
                4, 11, 2, 14, 15, 0, 8, 13, 3, 12, 9, 7, 5, 10, 6, 1,
                13, 0, 11, 7, 4, 9, 1, 10, 14, 3, 5, 12, 2, 15, 8, 6,
                1, 4, 11, 13, 12, 3, 7, 14, 10, 15, 6, 8, 0, 5, 9, 2,
                6, 11, 13, 8, 1, 4, 10, 7, 9, 5, 0, 15, 14, 2, 3, 12
        },
        {
                13, 2, 8, 4, 6, 15, 11, 1, 10, 9, 3, 14, 5, 0, 12, 7,
                1, 15, 13, 8, 10, 3, 7, 4, 12, 5, 6, 11, 0, 14, 9, 2,
                7, 11, 4, 1, 9, 12, 14, 2, 0, 6, 10, 13, 15, 3, 5, 8,
                2, 1, 14, 7, 4, 10, 8, 13, 15, 12, 9, 0, 3, 5, 6, 11
        }
};

__device__ int d_straight_permutation_table[32] = {
        16,  7, 20, 21,
        29, 12, 28, 17,
        1, 15, 23, 26,
        5, 18, 31, 10,
        2,  8, 24, 14,
        32, 27,  3,  9,
        19, 13, 30,  6,
        22, 11,  4, 25
};

__device__ int d_final_permutation_table[64] = {
        40, 8, 48, 16, 56, 24, 64, 32,
        39, 7, 47, 15, 55, 23, 63, 31,
        38, 6, 46, 14, 54, 22, 62, 30,
        37, 5, 45, 13, 53, 21, 61, 29,
        36, 4, 44, 12, 52, 20, 60, 28,
        35, 3, 43, 11, 51, 19, 59, 27,
        34, 2, 42, 10, 50, 18, 58, 26,
        33, 1, 41, 9, 49, 17, 57, 25
};

__device__ int* cuda_des_encrypt_text(int* bin_plain_text, int* sub_keys, int* cipher_text, int blockSize, int threads_number,
    int* result_initial_permutation, int* left_block, int* right_block, int* right_expanded, int* xor_result, 
    int* block, int* s_box_result, int* s_box_permuted_result, int* new_left_block, int* combined_key) {
    
    unsigned int index = blockIdx.x * blockSize + threadIdx.x;
    
    for (int i = 0; i < 64; i++) {
        result_initial_permutation[index * 64 + i] = bin_plain_text[index * 64 + d_initial_permutation_table[i] - 1]; 
    }

    for (int i = 0; i < 32; i++) {
        left_block[index * 32 + i] = result_initial_permutation[index * 64 + i];
        right_block[index * 32 + i] = result_initial_permutation[index * 64 + 32 + i];
    }

    for (int round = 0; round < 16; round ++) {
        for (int i = 0; i < 48; i++) {
            right_expanded[index * 48 + i] = right_block[index * 32 + d_expansion_permutation_table[i] - 1];
        }

        for (int i = 0; i < 48; i++) {
            xor_result[index * 48 + i] = right_expanded[index * 48 + i] ^ sub_keys[round * 48 + i];
        }

        for (int i = 0; i < 8; i++) {
            int row = (xor_result[index * 48 + i * 6] << 1) + xor_result[index * 48 + i * 6 + 5];
            int col = (xor_result[index * 48 + i * 6 + 1] << 3) + (xor_result[index * 48 + i * 6 + 2] << 2) + (xor_result[index * 48 + i * 6 + 3] << 1) + xor_result[index * 48 + i * 6 + 4];
            int val = d_s_box_table[i][row][col];
            for (int j = 0; j < 4; j++) {
                s_box_result[index * 32 + i * 4 + j] = (val >> (3 - j)) & 1;
            }
        }

        for (int i = 0; i < 32; i++) {
            s_box_permuted_result[index * 32 + i] = s_box_result[index * 32 + d_straight_permutation_table[i] - 1];
        }

        for (int i = 0; i < 32; i++) {
            new_left_block[index * 32 + i] = left_block[index * 32 + i] ^ s_box_permuted_result[index * 32 + i];
        }

        if (round != 15) {
            for (int i = 0; i < 32; i++) {
                left_block[index * 32 + i] = right_block[index * 32 + i];
                right_block[index * 32 + i] = new_left_block[index * 32 + i];
            }
        } else {
            for (int i = 0; i < 32; i++) {
                left_block[index * 32 + i] = new_left_block[index * 32 + i];
            }
        }
    }
    
    for (int i = 0; i< 32; i++) {
        combined_key[index * 64 + i] = left_block[index * 32 + i];
        combined_key[index * 64 + 32 + i] = right_block[index * 32 + i];
    }

    for (int i = 0; i < 64; i++) {
        cipher_text[index * 64 + i] = combined_key[index * 64 + d_final_permutation_table[i] - 1];
    }

    return cipher_text;
}


__device__ char* generate_all_possible_password(char* password, int password_length, int blockSize, long iteration) {
    unsigned int index = blockIdx.x * blockSize + threadIdx.x;
    
    for (int j = 0; j < password_length; j++) {
        password[index * password_length + j] = d_allowed_char[iteration % d_allowed_char_size];
        iteration /= d_allowed_char_size;
    } 

    return password;
}

__device__ int* d_string_to_binary(char* string_text, int password_length, int* bin_text, int blockSize) {
    unsigned int index = blockIdx.x * blockSize + threadIdx.x;
    
    int current_index = 0;
    for (int i = 0; i < password_length; i++) {
        char currentChar = string_text[index * password_length + i];
        for (int j = 7; j >= 0; --j) {
            bin_text[index * 64 + current_index++] = (currentChar >> j) & 1;
        }
    }

    return bin_text;
}

__device__ bool isBinaryEqual(int* string1, int* string2, int blockSize){
    unsigned int index = blockIdx.x * blockSize + threadIdx.x;

    bool isEqual = true;
    for (int i = 0; i < 64; i++) {
        if(string1[(index * 64) + i] != string2[i]){
            isEqual = false;
        }
    }

    return isEqual;
}

__global__ void brute_force_attack(int* cipher_password_target, int* d_sub_keys, int blockSize, int threads_number, 
    int password_length, char* current_password, int* bin_current_password, int* cipher_current_password,
    int* result_initial_permutation, int* left_block, int* right_block, int* right_expanded, int* xor_result, 
    int* block, int* s_box_result, int* s_box_permuted_result, int* new_left_block, int* combined_key) {
    
    unsigned int index = blockIdx.x * blockSize + threadIdx.x;

    long number_of_possible_passwords = (long)pow((double)d_allowed_char_size,(double)(password_length));
    for(long i = index; i < number_of_possible_passwords; i += threads_number){
        //printf("iteration %d \n", i);
        generate_all_possible_password(current_password, password_length, blockSize, i);
        d_string_to_binary(current_password, password_length, bin_current_password, blockSize);
        cuda_des_encrypt_text(bin_current_password, d_sub_keys, cipher_current_password, blockSize, threads_number,
         result_initial_permutation, left_block, right_block, right_expanded, xor_result, 
         block, s_box_result, s_box_permuted_result, new_left_block, combined_key);
    
        if(isBinaryEqual(cipher_current_password, cipher_password_target, blockSize)){
            printf("Find by thread n. %d! \n", index);
            printf("password is: '");
            for(int j = 0; j < 8; j++){
                printf("%c", current_password[index * password_length + j]);
            }
            printf("' in DES: '");
            for(int j = 0; j < 64; j++){
                printf("%d", cipher_current_password[index * password_length * 8 + j]);
            }
            printf("'\n");
            __trap();
            return;
        }
    }     
}

void setupGrid(int threads_number, int blockSize, int *num_block, int *threads_per_block) {
    if (threads_number <= blockSize) {
        *threads_per_block = threads_number;
        *num_block = 1;
    } else if ((threads_number % 256) == 0) {
        *threads_per_block = 256;
        *num_block = threads_number / 256;
        blockSize = 256;
    } else if ((threads_number % 128) == 0) {
        *threads_per_block = 128;
        *num_block = threads_number / 128;
        blockSize = 128;
    } else if ((threads_number % 32) == 0) {
        *threads_per_block = 32;
        *num_block = threads_number / 32;
        blockSize = 32;
    } else {
        int mod256 = threads_number % 256;
        int mod128 = threads_number % 128;
        int mod32 = threads_number % 32;

        if (mod256 <= 128 && mod256 <= mod128 && mod256 <= mod32) {
            *threads_per_block = threads_number + (32 - mod256);
            blockSize = 256;
        } else if (mod128 <= mod32) {
            *threads_per_block = threads_number + (128 - mod128);
            blockSize = 128;
        } else {
            *threads_per_block = threads_number + (32 - mod32);
            blockSize = 32;
        }
        
        *num_block = threads_number / *threads_per_block;
    }
}

void getGPUProperties(int gpuID) {
    hipDeviceProp_t prop;
    int deviceId = gpuID;

    hipGetDeviceProperties(&prop, deviceId);

    std::cout << "GPU info:" << std::endl;
    std::cout << "Name: " << prop.name << std::endl;
    std::cout << "Max number of threads for block: " << prop.maxThreadsPerBlock << std::endl;
    std::cout << "Max blocks number : " << prop.maxGridSize[0] << std::endl;
}

int main() {
    //SETUP DES
    const char* key = "A4rT9v.w";
    int* des_key = (int*) malloc(64 * sizeof(int));
    string_to_binary(key, 8, des_key);

    cout << "Binary representation of the key '" << key << "': ";
    for(int i = 0; i < 64; i++){
        cout << des_key[i];
    }
    cout << endl;
    
    int** sub_keys = create_sub_keys(des_key);
    for(int i = 0; i < 16; i++){
        for(int j = 0; j < 48; j++){
        }
    }

    int d_sub_keys_1d[16 * 48];
    for (int i = 0; i < 16; i++) {
        for (int j = 0; j < 48; j++) {
            d_sub_keys_1d[(i * 48) + j] = sub_keys[i][j];  
        }
    }
    
    cout << endl;

    //SETUP TARGET PASSWORD
    const char* password = "2/W.caaa";
    int* cipher_password_target = des_encrypt_text(password, d_sub_keys_1d);
    cout << "Password to find: '" << password << "' encrypted with DES: ";
    for(int i = 0; i < 64; i++){
        cout << cipher_password_target[i];
    }

    cout << endl;

    //SETUP CUDA
    //getGPUProperties(0); //Get GPU info
    unsigned int threads_number =384;
    int blockSize = 32;

    int threads_per_block;
    int num_block;
    setupGrid(threads_number, blockSize, &num_block, &threads_per_block);
    printf("Setup: <<<Grid Size: %d, Threads per Block: %d>>>\n", num_block, threads_per_block);

    int password_length = 8;
    char* current_password;
    int* bin_current_password;
    int* cipher_current_password;
    int* result_initial_permutation;
    int* left_block;
    int* right_block;
    int* right_expanded;
    int* xor_result;
    int* block; 
    int* s_box_result;
    int* s_box_permuted_result;
    int* new_left_block;
    int* combined_key;

    hipMalloc((void**)&current_password, (threads_number * password_length * sizeof(char)));
    hipMalloc((void**)&bin_current_password, (threads_number * password_length * 8 * sizeof(int)));
    hipMalloc((void**)&cipher_current_password, (threads_number * 64 * sizeof(int)));
    hipMalloc((void**)&result_initial_permutation, (threads_number * 64 * sizeof(int)));
    hipMalloc((void**)&left_block, (threads_number * 32 * sizeof(int)));
    hipMalloc((void**)&right_block, (threads_number * 32 * sizeof(int)));
    hipMalloc((void**)&right_expanded, (threads_number * 48 * sizeof(int)));
    hipMalloc((void**)&xor_result, (threads_number * 48 * sizeof(int)));
    hipMalloc((void**)&block, (threads_number * 6 * sizeof(int)));
    hipMalloc((void**)&s_box_result, (threads_number * 32 * sizeof(int)));
    hipMalloc((void**)&s_box_permuted_result, (threads_number * 32 * sizeof(int)));
    hipMalloc((void**)&new_left_block, (threads_number * 32 * sizeof(int)));
    hipMalloc((void**)&combined_key, (threads_number * 64 * sizeof(int)));

    int* d_cipher_password_target;
    hipMalloc((void**)&d_cipher_password_target, (threads_number * password_length * 8 * sizeof(int)));
    hipMemcpy(d_cipher_password_target, cipher_password_target, password_length * 8 * sizeof(int), hipMemcpyHostToDevice);

    int* d_sub_keys;
    hipMalloc((void**)&d_sub_keys, (16 * 48 * sizeof(int)));
    hipMemcpy(d_sub_keys, d_sub_keys_1d, 16 * 48 * sizeof(int), hipMemcpyHostToDevice);

    //DA SISTEMARE --> è possibile avere qualcosa utilizzabile sia in gpu che in cpu ? tanto son odat statici
    long number_of_possible_passwords = (long)pow((double)allowed_char_size,(double)(password_length));
    cout << "Total of possible password: " << number_of_possible_passwords << " with: " << password_length << " characters" << endl;
    cout << endl;
    cout << "Brute force attack started ..." << endl;
    struct timeval start_time, end_time;
    gettimeofday(&start_time, NULL);

    cout << endl;
    
    brute_force_attack<<<num_block, threads_per_block>>>(d_cipher_password_target, d_sub_keys, blockSize, threads_number, 
        password_length, current_password, bin_current_password, 
        cipher_current_password, result_initial_permutation, left_block, right_block, right_expanded, xor_result, 
        block, s_box_result, s_box_permuted_result, new_left_block, combined_key);
    hipDeviceSynchronize();

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        cout << "CUDA Error: " << hipGetErrorString(error) << endl;
        cout << "Brute force attack aborted" << endl;
    }
    
    cout << endl;

    gettimeofday(&end_time, NULL);
    double total_time = ((end_time.tv_sec  - start_time.tv_sec) * 1000000u + end_time.tv_usec - start_time.tv_usec) / 1.e6;
    cout << "Execution Time: " << total_time << " s" << endl;
    cout << "Brute force attack terminated" << endl;

    hipFree(current_password);
    hipFree(bin_current_password);
    hipFree(cipher_current_password);
    hipFree(result_initial_permutation);
    hipFree(left_block);
    hipFree(right_block);
    hipFree(right_expanded);
    hipFree(xor_result);
    hipFree(block);
    hipFree(s_box_result);
    hipFree(s_box_permuted_result);
    hipFree(new_left_block);
    hipFree(combined_key);
    hipFree(d_sub_keys);
    hipFree(d_cipher_password_target);

    return 0;
}
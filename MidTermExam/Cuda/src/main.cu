#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "desKeyGenerator.h"
#include "desTextEncryptor.h"
#include "utility.h"

using namespace std;

__device__ int d_initial_permutation_table[64] = {
        58, 50, 42, 34, 26, 18, 10, 2,
        60, 52, 44, 36, 28, 20, 12, 4,
        62, 54, 46, 38, 30, 22, 14, 6,
        64, 56, 48, 40, 32, 24, 16, 8,
        57, 49, 41, 33, 25, 17,  9, 1,
        59, 51, 43, 35, 27, 19, 11, 3,
        61, 53, 45, 37, 29, 21, 13, 5,
        63, 55, 47, 39, 31, 23, 15, 7
};

__device__ int d_expansion_permutation_table[48] = {
        32, 1, 2, 3, 4, 5, 4, 5,
        6, 7, 8, 9, 8, 9, 10, 11,
        12, 13, 12, 13, 14, 15, 16, 17,
        16, 17, 18, 19, 20, 21, 20, 21,
        22, 23, 24, 25, 24, 25, 26, 27,
        28, 29, 28, 29, 30, 31, 32, 1
};

__device__ int d_s_box_table[8][4][16] = {
        {
                14, 4, 13, 1, 2, 15, 11, 8, 3, 10, 6, 12, 5, 9, 0, 7,
                0, 15, 7, 4, 14, 2, 13, 1, 10, 6, 12, 11, 9, 5, 3, 8,
                4, 1, 14, 8, 13, 6, 2, 11, 15, 12, 9, 7, 3, 10, 5, 0,
                15, 12, 8, 2, 4, 9, 1, 7, 5, 11, 3, 14, 10, 0, 6, 13
        },
        {
                15, 1, 8, 14, 6, 11, 3, 4, 9, 7, 2, 13, 12, 0, 5, 10,
                3, 13, 4, 7, 15, 2, 8, 14, 12, 0, 1, 10, 6, 9, 11, 5,
                0, 14, 7, 11, 10, 4, 13, 1, 5, 8, 12, 6, 9, 3, 2, 15,
                13, 8, 10, 1, 3, 15, 4, 2, 11, 6, 7, 12, 0, 5, 14, 9
        },
        {
                10, 0, 9, 14, 6, 3, 15, 5, 1, 13, 12, 7, 11, 4, 2, 8,
                13, 7, 0, 9, 3, 4, 6, 10, 2, 8, 5, 14, 12, 11, 15, 1,
                13, 6, 4, 9, 8, 15, 3, 0, 11, 1, 2, 12, 5, 10, 14, 7,
                1, 10, 13, 0, 6, 9, 8, 7, 4, 15, 14, 3, 11, 5, 2, 12
        },
        {
                7, 13, 14, 3, 0, 6, 9, 10, 1, 2, 8, 5, 11, 12, 4, 15,
                13, 8, 11, 5, 6, 15, 0, 3, 4, 7, 2, 12, 1, 10, 14, 9,
                10, 6, 9, 0, 12, 11, 7, 13, 15, 1, 3, 14, 5, 2, 8, 4,
                3, 15, 0, 6, 10, 1, 13, 8, 9, 4, 5, 11, 12, 7, 2, 14
        },
        {
                2, 12, 4, 1, 7, 10, 11, 6, 8, 5, 3, 15, 13, 0, 14, 9,
                14, 11, 2, 12, 4, 7, 13, 1, 5, 0, 15, 10, 3, 9, 8, 6,
                4, 2, 1, 11, 10, 13, 7, 8, 15, 9, 12, 5, 6, 3, 0, 14,
                11, 8, 12, 7, 1, 14, 2, 13, 6, 15, 0, 9, 10, 4, 5, 3
        },
        {
                12, 1, 10, 15, 9, 2, 6, 8, 0, 13, 3, 4, 14, 7, 5, 11,
                10, 15, 4, 2, 7, 12, 9, 5, 6, 1, 13, 14, 0, 11, 3, 8,
                9, 14, 15, 5, 2, 8, 12, 3, 7, 0, 4, 10, 1, 13, 11, 6,
                4, 3, 2, 12, 9, 5, 15, 10, 11, 14, 1, 7, 6, 0, 8, 13
        },
        {
                4, 11, 2, 14, 15, 0, 8, 13, 3, 12, 9, 7, 5, 10, 6, 1,
                13, 0, 11, 7, 4, 9, 1, 10, 14, 3, 5, 12, 2, 15, 8, 6,
                1, 4, 11, 13, 12, 3, 7, 14, 10, 15, 6, 8, 0, 5, 9, 2,
                6, 11, 13, 8, 1, 4, 10, 7, 9, 5, 0, 15, 14, 2, 3, 12
        },
        {
                13, 2, 8, 4, 6, 15, 11, 1, 10, 9, 3, 14, 5, 0, 12, 7,
                1, 15, 13, 8, 10, 3, 7, 4, 12, 5, 6, 11, 0, 14, 9, 2,
                7, 11, 4, 1, 9, 12, 14, 2, 0, 6, 10, 13, 15, 3, 5, 8,
                2, 1, 14, 7, 4, 10, 8, 13, 15, 12, 9, 0, 3, 5, 6, 11
        }
};

__device__ int d_straight_permutation_table[32] = {
        16,  7, 20, 21,
        29, 12, 28, 17,
        1, 15, 23, 26,
        5, 18, 31, 10,
        2,  8, 24, 14,
        32, 27,  3,  9,
        19, 13, 30,  6,
        22, 11,  4, 25
};

__device__ int d_final_permutation_table[64] = {
        40, 8, 48, 16, 56, 24, 64, 32,
        39, 7, 47, 15, 55, 23, 63, 31,
        38, 6, 46, 14, 54, 22, 62, 30,
        37, 5, 45, 13, 53, 21, 61, 29,
        36, 4, 44, 12, 52, 20, 60, 28,
        35, 3, 43, 11, 51, 19, 59, 27,
        34, 2, 42, 10, 50, 18, 58, 26,
        33, 1, 41, 9, 49, 17, 57, 25
};

__device__ char* cuda_des_encrypt_text(char* bin_plain_text, char* sub_keys, char* cipher_text, int blockSize, int threads_number,
    char* result_initial_permutation, char* left_block, char* right_block, char* right_expanded, char* xor_result, 
    char* block, char* s_box_result, char* s_box_permuted_result, char* new_left_block, char* combined_key) {

    int index = blockIdx.x * blockSize + threadIdx.x;

    for (int i = 0; i < 64; i++) {
        result_initial_permutation[index * 64 + i] = bin_plain_text[d_initial_permutation_table[i] - 1];
    }

    for (int i = 0; i < 32; i++) {
        left_block[index * 32 + i] = result_initial_permutation[index * 64 + i];
        right_block[index * 32 + i] = result_initial_permutation[index * 64 + 32 + i];
    }

    for (int round = 0; round < 16; round ++) {
        for (int i = 0; i < 48; i++) {
            right_expanded[index * 48 + i] = right_block[index * 32 + d_expansion_permutation_table[i] - 1];
        }

        for (int i = 0; i < 48; i++) {
            xor_result[index * 48 + i] = sub_keys[round * 48 + i] ^ (right_expanded[index * 48 + i] - '0');
        }

        for (int i = 0; i < 8; i++){
            for (int j = 0; j < 6; j++){
                block[j] = xor_result[index * 48 + i * 6 + j];
            }

            int row = (block[0] - '0') * 2 + (block[5] - '0');
            int col = (block[1] - '0') * 8 + (block[2] - '0') * 4 + (block[3] - '0') * 2 + (block[4] - '0');

            int s_box_value = d_s_box_table[i][row][col];
            
            for (int k = 0; k < 4; k++) {
                s_box_result[index * 32 + i * 4 + k] = ((s_box_value >> (3 - k)) & 1) + '0';
            }
        }

        for (int i = 0; i < 32; i++) {
            s_box_permuted_result[index * 32 + i] = s_box_result[index * 32 + d_straight_permutation_table[i] - 1];
        }

        for (int i = 0; i < 32; i++) {
            new_left_block[index * 32 + i] = (left_block[index * 32 + i] - '0') ^ (s_box_permuted_result[index * 32 + i] - '0') + '0';
        }

        if (round != 15) {
            for (int i = 0; i < 32; i++) {
                left_block[index * 32 + i] = right_block[index * 32 + i];
                right_block[index * 32 + i] = new_left_block[index * 32 + i];
            }
        } else {
            for (int i = 0; i < 32; i++) {
                left_block[index * 32 + i] = new_left_block[index * 32 + i];
            }
        }
    }
    
    for (int i = 0; i< 32; i++) {
        combined_key[index * 64 + i] = left_block[index * 32 + i];
        combined_key[index * 64 + 32 + i] = right_block[index * 32 + i];
    }

    for (int i = 0; i < 64; i++) {
        cipher_text[index * 64 + i] = combined_key[index * 64 + d_final_permutation_table[i] - 1];
    }

    return cipher_text;
}

__device__ char* generate_all_possible_password(char* password, int password_length, int blockSize, long iteration) {
    unsigned int index = blockIdx.x * blockSize + threadIdx.x;
    for (int j = 0; j < password_length - 1; j++) {
        password[index * password_length + j] = d_allowed_char[iteration % allowed_char_size];
        iteration /= allowed_char_size;
    } 

    //JUST FOR TESTING
    /*
    char prev[8];
    for(int i=0; i< 7; i++){
        prev[i] = password[index*password_length + i];
    } 
    printf("thread n. %d pwd: %s\n",index, prev);
    */

    return password;
}

__device__ char* d_string_to_binary(char* string_text, int password_length, char* bin_text, int blockSize) {
    unsigned int index = blockIdx.x * blockSize + threadIdx.x;

    int start_index = 0;
    for (int i = 0; i < password_length - 1; i++) {
        for (int j = 7; j >= 0; --j) {
            bin_text[(index * password_length * 8) + (i * 8) + (7 - j)] = ((string_text[i] >> j) & 1) ? '1' : '0';
        }
    }

    return bin_text;
}

__device__ bool isBinaryStringEqual(char* string1, char* string2, int blockSize){
    unsigned int index = blockIdx.x * blockSize + threadIdx.x;
    bool isEqual = true;
    for (int i= 0; i < 63; i++) {
        if(string1[(index * 64) + i] != string2[i]){
            isEqual = false;
        }
    }

    return isEqual;
}

__global__ void brute_force_attack(char* cipher_password_target, char* sub_keys_1d, int blockSize, int threads_number, 
    int password_length, char* current_password, char* bin_current_password, char* cipher_current_password,
    char* result_initial_permutation, char* left_block, char* right_block, char* right_expanded, char* xor_result, 
    char* block, char* s_box_result, char* s_box_permuted_result, char* new_left_block, char* combined_key) {
    
    unsigned int index = blockIdx.x * blockSize + threadIdx.x;
    long number_of_possible_passwords = (long)pow((double)allowed_char_size,(double)password_length);
    bool password_found = false;
    
    long password_per_thread = number_of_possible_passwords / threads_number;
    long start_index = index * password_per_thread;
    long end_index = password_per_thread + start_index;
    
    for (long i = start_index; i < end_index; i++){
        generate_all_possible_password(current_password, password_length, blockSize, i);
        d_string_to_binary(current_password, password_length, bin_current_password, blockSize);
        cuda_des_encrypt_text(bin_current_password, sub_keys_1d, cipher_current_password, blockSize, threads_number,
            result_initial_permutation, left_block, right_block, right_expanded, xor_result, 
            block, s_box_result, s_box_permuted_result, new_left_block, combined_key);
        
        if (isBinaryStringEqual(cipher_current_password, cipher_password_target, blockSize)){
            password_found = true;
            printf("Password is found! \n");
            return;
        }
        

    } 
}

void getGPUProperties(int gpuID) {
    hipDeviceProp_t prop;
    int deviceId = gpuID; // ID della tua GPU, potrebbe essere diverso se hai più di una GPU installata

    hipGetDeviceProperties(&prop, deviceId);

    std::cout << "Proprietà della GPU:" << std::endl;
    std::cout << "Nome: " << prop.name << std::endl;
    std::cout << "Massimo numero di thread per blocco: " << prop.maxThreadsPerBlock << std::endl;
}

int main() {

    //SETUP DES
    const char* key = "A4rT9v.w";
    char* des_key = (char*) malloc(64 * sizeof(char));
    string_to_binary(key, des_key);

    cout << "Binary representation of the key'" << key << "': " << des_key << endl;
    
    char** sub_keys = create_sub_keys(des_key);
    char sub_keys_1d[16 * 48]; //creiamo un array 1D in cui inseriamo le chiavi da trasferire in GPU
    for (int i = 0; i < 16; i++) {
        for (int j = 0; j < 48; j++) {
            sub_keys_1d[(i * 48) + j] = sub_keys[i][j];  
        }
    }

    cout << endl;

    //SETUP TARGET PASSWORD
    const char* password = "Zaaaaaaa";
    char* cipher_password_target = des_encrypt_text(password, sub_keys_1d);
    cout << "Password to find: '" << password << "' encrypted with DES: " << cipher_password_target << endl;

    //SETUP CUDA
    //getGPUProperties(0); //Get info
    unsigned int threads_number = 1;
    int blockSize = 32;

    //Setup block
    int numBlocks;
    int threads_per_block;
    if (threads_number <= 32) {
        numBlocks = 1;
        threads_per_block = threads_number;
    } else if (threads_number % 128 == 0) {
        numBlocks = threads_number / 128;
        threads_per_block = 128;
    } else {
        numBlocks = (threads_number + 31) / 32;  // Arrotonda al prossimo multiplo di 32
        threads_per_block = 32;
    }
    printf("Setup: <<<%d,%d>>>\n", numBlocks, threads_per_block);
    
    int password_length = 8;  //length + 1 for \0
    char* current_password;
    char* bin_current_password;
    char* cipher_current_password;
    char* result_initial_permutation;
    char* left_block;
    char* right_block;
    char* right_expanded;
    char* xor_result;
    char* block; 
    char* s_box_result;
    char* s_box_permuted_result;
    char* new_left_block;
    char* combined_key;

    hipMalloc((void**)&current_password, (threads_number * (password_length +1) * sizeof(char)));
    hipMalloc((void**)&bin_current_password, (threads_number * 64 * sizeof(char)));
    hipMalloc((void**)&cipher_current_password, (threads_number * 64 * sizeof(char)));
    hipMalloc((void**)&result_initial_permutation, (threads_number * 64 * sizeof(char)));
    hipMalloc((void**)&left_block, (threads_number * 32 * sizeof(char)));
    hipMalloc((void**)&right_block, (threads_number * 32 * sizeof(char)));
    hipMalloc((void**)&right_expanded, (threads_number * 48 * sizeof(char)));
    hipMalloc((void**)&xor_result, (threads_number * 48 * sizeof(char)));
    hipMalloc((void**)&block, (threads_number * 6 * sizeof(char)));
    hipMalloc((void**)&s_box_result, (threads_number * 32 * sizeof(char)));
    hipMalloc((void**)&s_box_permuted_result, (threads_number * 32 * sizeof(char)));
    hipMalloc((void**)&new_left_block, (threads_number * 32 * sizeof(char)));
    hipMalloc((void**)&combined_key, (threads_number * 64 * sizeof(char)));

    char* d_cipher_password_target;
    hipMalloc((void**)&d_cipher_password_target, (threads_number * 64 * sizeof(char)));
    hipMemcpy(d_cipher_password_target, cipher_password_target, 64 * sizeof(char), hipMemcpyHostToDevice);

    char* d_sub_keys;
    hipMalloc((void**)&d_sub_keys, (16 * 48 * sizeof(char)));
    hipMemcpy(d_sub_keys, sub_keys_1d, 16 * 48 * sizeof(char), hipMemcpyHostToDevice);

    long number_of_possible_passwords = (long)pow((double)allowed_char_size,(double)(password_length));
    cout << "Total of possible password: " << number_of_possible_passwords << " with: " << password_length << " characters" << endl;
    cout << endl;
    cout << "Brute force attack started ..." << endl;
    struct timeval start_time, end_time;
    gettimeofday(&start_time, NULL);

    cout << endl;

    brute_force_attack<<<numBlocks, threads_per_block>>>(d_cipher_password_target, d_sub_keys, blockSize, threads_number, 
        (password_length +1), current_password, bin_current_password, 
        cipher_current_password, result_initial_permutation, left_block, right_block, right_expanded, xor_result, 
        block, s_box_result, s_box_permuted_result, new_left_block, combined_key);
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        cout << "Errore CUDA: " << hipGetErrorString(error) << endl;
    }

    cout << endl;

    gettimeofday(&end_time, NULL);
    double total_time = ((end_time.tv_sec  - start_time.tv_sec) * 1000000u + end_time.tv_usec - start_time.tv_usec) / 1.e6;
    cout << "Tempo richiesto: " << total_time << " s" << endl;
    cout << "Attacco brute force terminato" << endl;


    hipFree(current_password);
    hipFree(bin_current_password);
    hipFree(cipher_current_password);
    hipFree(result_initial_permutation);
    hipFree(left_block);
    hipFree(right_block);
    hipFree(right_expanded);
    hipFree(xor_result);
    hipFree(block);
    hipFree(s_box_result);
    hipFree(s_box_permuted_result);
    hipFree(new_left_block);
    hipFree(combined_key);
    hipFree(d_sub_keys);
    hipFree(d_cipher_password_target);

    return 0;
}
